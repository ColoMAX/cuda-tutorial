#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6

double gettime(){
    struct timeval t;
    gettimeofday(&t, 0);
    return (t.tv_sec * 1000000 + t.tv_usec);
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for(int i = 0; i < n; i ++){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Timing execution time 
    double timer = -gettime();
    
    vector_add<<<1,1>>>(d_out, d_a, d_b, N);
    
    hipDeviceSynchronize();
    timer += gettime();

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for(int i = 0; i < N; i++){
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }

    printf("PASSED: %f s\n", timer / 1e6);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    free(a); 
    free(b); 
    free(out);
}
