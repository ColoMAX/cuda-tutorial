#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void cudaHello(){
    printf("Hello World with NVCC!\n");
}

int main() {
    cudaHello<<<1,1>>>(); 
    return 0;
}
